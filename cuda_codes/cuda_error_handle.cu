
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void my_cudasafe( hipError_t error, char* message)
{
	if(error!=hipSuccess) 
	{ 
		fprintf(stderr,"ERROR: %s : %i\n",message,error); 
		exit(-1); 
	}
}

__global__ void matrixMulKernel(float *md, float *nd, float *pd, int width)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float Pvalue = 0;

	for(int k=0; k < width; ++k)
	{
		float Mdelement = md[ty*width+k];
		float Ndelement = nd[k*width+tx];
		Pvalue += Mdelement*Ndelement; 
	}

	pd[ty * width + tx] = Pvalue;
}

void matrixmul(float *m, float *n, float *p, int width)
{
	int size = width*width*sizeof(float);
	float *md, *nd, *pd;
	

	my_cudasafe(hipMalloc((void **)&md, size),"Cuda malloc : md");
	my_cudasafe(hipMemcpy(md, m, size, hipMemcpyHostToDevice),"Cuda memcopy : md");

	my_cudasafe(hipMalloc((void **)&nd, size),"Cuda malloc : nd");
	my_cudasafe(hipMemcpy(nd, n, size, hipMemcpyHostToDevice),"Cuda memcopy : nd");

	my_cudasafe(hipMalloc((void **)&pd, size),"Cuda malloc : pd");

	dim3 dimBlock(width,width);
	dim3 dimGrid(1,1);



	matrixMulKernel<<<dimGrid,dimBlock>>>(md,nd,pd,width);
	

	my_cudasafe(hipMemcpy(p, pd, size, hipMemcpyDeviceToHost),"Cuda memcopy : pd");
	hipFree(md);
	hipFree(nd);
	hipFree(pd);
}


int main()
{
	float *m, *n, *p;
	int width = 3;
	int  i = 0, j= 0;

	m = (float *)malloc(width*width*sizeof(float));
	n = (float *)malloc(width*width*sizeof(float));
	p = (float *)malloc(width*width*sizeof(float));

	for(i=0; i< width; i++)
	{
		for(j=0; j< width; j++)
		{
			m[width*i+j] = 1.00;
			n[width*i+j] = 1.00;
		}
	}

	matrixmul(m,n,p,width);

	printf("\n\n M : \n");
	for(i=0; i< width; i++)
	{
		for(j=0; j< width; j++)
		{
			printf("%f ",m[width*i+j]);
		}
		printf("\n");
	}
	printf("\n\n N : \n");
	for(i=0; i< width; i++)
	{
		for(j=0; j< width; j++)
		{
			printf("%f ",n[width*i+j]);
		}
		printf("\n");
	}
	printf("\n\n P : \n");
	for(i=0; i< width; i++)
	{
		for(j=0; j< width; j++)
		{
			printf("%f ",p[width*i+j]);
		}
		printf("\n");
	}
	return 0;
}
